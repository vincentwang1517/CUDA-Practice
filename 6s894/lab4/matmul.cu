#include <chrono>
#include <cstdint>
#include <cstdio>
#include <hip/hip_runtime.h>
#include <fstream>
#include <iostream>
#include <random>
#include <utility>
#include <vector>

#define TILE_SIZE (16)

void cuda_check(hipError_t code, const char *file, int line) {
    if (code != hipSuccess) {
        std::cerr << "CUDA error at " << file << ":" << line << ": "
                  << hipGetErrorString(code) << std::endl;
        exit(1);
    }
}

#define CUDA_CHECK(x) \
    do { \
        cuda_check((x), __FILE__, __LINE__); \
    } while (0)

////////////////////////////////////////////////////////////////////////////////
// CPU Reference Implementation (Too slow to actually run!)
//
// void matmul_cpu_naive(
//     int32_t size_i,
//     int32_t size_j,
//     int32_t size_k,
//     float const *a,
//     float const *b,
//     float *c) {
//     for (int32_t i = 0; i < size_i; ++i) {
//         for (int32_t j = 0; j < size_j; ++j) {
//             float sum = 0.0;
//             for (int32_t k = 0; k < size_k; ++k) {
//                 sum += a[i * size_k + k] * b[k * size_j + j];
//             }
//             c[i * size_j + j] = sum;
//         }
//     }
// }

/// <--- your code here --->

////////////////////////////////////////////////////////////////////////////////
// GPU Implementation (With Reuse in L1/Shmem)

namespace matmul_l1 {

__global__ void matmul_l1(
    int32_t size_i,
    int32_t size_j,
    int32_t size_k,
    float const *a,
    float const *b,
    float *c) {
    /* TODO: your GPU code here */

    // smem
    __shared__ float tile_a[TILE_SIZE][TILE_SIZE];
    __shared__ float tile_b[TILE_SIZE][TILE_SIZE];

    // index
    int tx = threadIdx.x;
    int ty = threadIdx.y;
    int row = blockIdx.y * blockDim.y + threadIdx.y; // global row index
    int col = blockIdx.x * blockDim.x + threadIdx.x; // global column index
    float value = 0.0f;

    for (int t = 0; t < size_k/TILE_SIZE; t++) {
        // load smem
        tile_a[ty][tx] = a[row * size_k + t * TILE_SIZE + tx];
        tile_b[ty][tx] = b[(t * TILE_SIZE + ty) * size_j + col];
        __syncthreads();

        // FMA
        for (int k = 0; k < TILE_SIZE; k++) {
            value += tile_a[ty][k] * tile_b[k][tx];
        }
        __syncthreads();
    }

    // assign value
    if (row < size_i && col < size_j) {
        c[row * size_j + col] = value;
    }
}

void launch_matmul_l1(
    int32_t size_i,
    int32_t size_j,
    int32_t size_k,
    float const *a,
    float const *b,
    float *c) {
    /* TODO: your CPU code here */

    dim3 block_size = dim3(TILE_SIZE, TILE_SIZE);
    dim3 grid_size = dim3((size_i + TILE_SIZE - 1) / TILE_SIZE, (size_j + TILE_SIZE -1) / TILE_SIZE);

    matmul_l1<<<grid_size, block_size>>>(size_i, size_j, size_k, a, b, c);
}

}; // namespace matmul_l1

////////////////////////////////////////////////////////////////////////////////
// GPU Implementation (With Reuse in L1/Shmem and Registers)

namespace matmul_l1_reg {

__global__ void matmul_l1_reg(
    int32_t size_i,
    int32_t size_j,
    int32_t size_k,
    float const *a,
    float const *b,
    float *c) {
    /* TODO: your GPU code here */
}

void launch_matmul_l1_reg(
    int32_t size_i,
    int32_t size_j,
    int32_t size_k,
    float const *a,
    float const *b,
    float *c) {
    /* TODO: your CPU code here */
}

}; // namespace matmul_l1_reg

/// <--- /your code here --->

////////////////////////////////////////////////////////////////////////////////
///          YOU DO NOT NEED TO MODIFY THE CODE BELOW HERE.                  ///
////////////////////////////////////////////////////////////////////////////////

std::vector<float> read_data(std::string const &path, int32_t size) {
    std::ifstream file(path, std::ios::binary);
    std::vector<float> data(size);
    file.read(reinterpret_cast<char *>(data.data()), data.size() * sizeof(float));
    if (file.fail()) {
        std::cerr << "Failed to read " << path << std::endl;
        std::abort();
    }
    return data;
}

template <typename F>
double benchmark_ms(double target_time_ms, int32_t num_iters_inner, F &&f) {
    double best_time_ms = std::numeric_limits<double>::infinity();
    double elapsed_ms = 0.0;
    while (elapsed_ms < target_time_ms) {
        CUDA_CHECK(hipDeviceSynchronize());
        auto start = std::chrono::high_resolution_clock::now();
        for (int32_t i = 0; i < num_iters_inner; ++i) {
            f();
        }
        CUDA_CHECK(hipDeviceSynchronize());
        auto end = std::chrono::high_resolution_clock::now();
        double this_ms = std::chrono::duration<double, std::milli>(end - start).count();
        elapsed_ms += this_ms;
        best_time_ms = std::min(best_time_ms, this_ms / num_iters_inner);
    }
    return best_time_ms;
}

struct BenchmarkResult {
    char const *name;
    double elapsed_ms;
};

struct BenchmarkConfig {
    int32_t size_i;
    int32_t size_j;
    int32_t size_k;
    bool save_result;
};

template <typename Impl>
void run_tests_for_size(
    std::string const &test_data_dir,
    std::vector<BenchmarkResult> &saved_results,
    std::vector<BenchmarkConfig> const &configs) {
    for (auto config : configs) {
        auto size_i = config.size_i;
        auto size_j = config.size_j;
        auto size_k = config.size_k;

        auto path_prefix = test_data_dir + "/test_" + std::to_string(size_i) + "x" +
            std::to_string(size_j) + "x" + std::to_string(size_k);
        auto a = read_data(path_prefix + "_a.bin", size_i * size_k);
        auto b = read_data(path_prefix + "_b.bin", size_k * size_j);
        auto c = read_data(path_prefix + "_c.bin", size_i * size_j);

        float *a_gpu;
        float *b_gpu;
        float *c_gpu;
        CUDA_CHECK(hipMalloc(&a_gpu, size_i * size_k * sizeof(float)));
        CUDA_CHECK(hipMalloc(&b_gpu, size_k * size_j * sizeof(float)));
        CUDA_CHECK(hipMalloc(&c_gpu, size_i * size_j * sizeof(float)));

        CUDA_CHECK(hipMemcpy(
            a_gpu,
            a.data(),
            size_i * size_k * sizeof(float),
            hipMemcpyHostToDevice));
        CUDA_CHECK(hipMemcpy(
            b_gpu,
            b.data(),
            size_k * size_j * sizeof(float),
            hipMemcpyHostToDevice));

        Impl::run(size_i, size_j, size_k, a_gpu, b_gpu, c_gpu);

        std::vector<float> c_out_host(size_i * size_j);
        CUDA_CHECK(hipMemcpy(
            c_out_host.data(),
            c_gpu,
            size_i * size_j * sizeof(float),
            hipMemcpyDeviceToHost));

        double mse = 0.0;
        double ref_mean_square = 0.0;
        for (int32_t i = 0; i < size_i; ++i) {
            for (int32_t j = 0; j < size_j; ++j) {
                float diff = c_out_host[i * size_j + j] - c[i * size_j + j];
                mse += diff * diff;
                ref_mean_square += c[i * size_j + j] * c[i * size_j + j];
            }
        }
        mse /= size_i * size_j;
        ref_mean_square /= size_i * size_j;
        float rmse = std::sqrt(mse);
        float rel_rmse = rmse / std::sqrt(ref_mean_square);

        printf("  size %4d * %4d * %4d:\n", size_i, size_j, size_k);
        printf("    correctness: %.02e relative RMSE\n", rel_rmse);

        if (rel_rmse > 1e-5) {
            printf("    skipping benchmark (incorrect)\n");
        } else {
            double elapsed_ms = benchmark_ms(1000.0, 4, [&]() {
                Impl::run(size_i, size_j, size_k, a_gpu, b_gpu, c_gpu);
            });

            printf("    run time: %6.02f ms\n", elapsed_ms);

            double tflop = 2.0 * size_i * size_k * size_j * 1e-12;
            printf("    throughput: %5.02f TFLOP/s\n", tflop / (elapsed_ms * 1e-3));

            if (config.save_result) {
                saved_results.push_back({Impl::name, elapsed_ms});
            }
        }

        printf("\n");
    }
}

template <typename Impl>
void run_all_tests(
    std::string const &test_data_dir,
    std::vector<BenchmarkResult> &saved_results) {
    printf("%s:\n\n", Impl::name);
    run_tests_for_size<Impl>(test_data_dir, saved_results, {{256, 256, 256, false}});
    run_tests_for_size<Impl>(test_data_dir, saved_results, {{3072, 3072, 3072, true}});
}

struct MatmulL1 {
    constexpr static char const *name = "matmul_l1";
    static void
    run(int32_t size_i,
        int32_t size_j,
        int32_t size_k,
        float const *a,
        float const *b,
        float *c) {
        matmul_l1::launch_matmul_l1(size_i, size_j, size_k, a, b, c);
    }
};

struct MatmulL1Reg {
    constexpr static char const *name = "matmul_l1_reg";
    static void
    run(int32_t size_i,
        int32_t size_j,
        int32_t size_k,
        float const *a,
        float const *b,
        float *c) {
        matmul_l1_reg::launch_matmul_l1_reg(size_i, size_j, size_k, a, b, c);
    }
};

int main(int argc, char **argv) {
    std::string test_data_dir = ".";
    if (char *c_str_test_data_dir = std::getenv("MATMUL_TEST_DATA_DIR")) {
        test_data_dir = c_str_test_data_dir;
    }

    auto saved_results = std::vector<BenchmarkResult>();

    run_all_tests<MatmulL1>(test_data_dir, saved_results);
    run_all_tests<MatmulL1Reg>(test_data_dir, saved_results);

    if (saved_results.size() > 1) {
        printf("speedups on largest problem size:\n");
        for (int32_t j = 1; j < saved_results.size(); ++j) {
            printf("\n");
            for (int32_t i = j; i > 0;) {
                --i;
                auto const &first = saved_results.at(i);
                auto const &second = saved_results.at(j);
                printf(
                    "  speedup %s -> %s: %.02fx\n",
                    first.name,
                    second.name,
                    first.elapsed_ms / second.elapsed_ms);
            }
        }
    }

    return 0;
}